#include "hip/hip_runtime.h"
#include "hopcroft_karp.c"

#define MAX_NODES 1000

/*
 * Kernel Function: cuda_bfs
 * -------------------------
 * Performs breadth-first search (BFS) on the bipartite graph represented as an adjacency matrix
 * to find augmenting paths for the Hopcroft-Karp algorithm using CUDA parallelism.
 * 
 * graph: Pointer to the adjacency matrix of the bipartite graph (device memory).
 * matchX: Array storing the matched vertices in set X (device memory).
 * matchY: Array storing the matched vertices in set Y (device memory).
 * queue: Array used as a queue for BFS traversal (device memory).
 * visited: Array to mark visited vertices during BFS traversal (device memory).
 * level: Array to store the level of each vertex during BFS traversal (device memory).
 * m: Pointer to the number of vertices in set X (device memory).
 * n: Pointer to the number of vertices in set Y (device memory).
 */
__global__ void cuda_bfs(int *graph, int *matchX, int *matchY, int *queue, int *visited, int *level, int *m, int *n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < *m) {
        int u = idx + 1;
        if (matchX[u] == -1) {
            level[u] = 0;
            queue[0] = u;
            int front = 0, rear = 1;
            while (front < rear) {
                int v = queue[front++];
                for (int i = 0; i < *n; ++i) {
                    if (graph[u * (*n) + i] && matchY[i + 1] == -1) {
                        matchX[u] = i + 1;
                        matchY[i + 1] = u;
                        return;
                    }
                    if (graph[u * (*n) + i] && level[matchY[i + 1]] == -1) {
                        level[matchY[i + 1]] = level[u] + 1;
                        queue[rear++] = matchY[i + 1];
                    }
                }
            }
        }
    }
}


/*
 * Function: hopcroftKarp_cuda
 * ----------------------------
 * Applies the Hopcroft-Karp algorithm for finding maximum matchings in a bipartite graph using CUDA.
 * 
 * graph: Pointer to the bipartite graph structure.
 * 
 * returns: The number of edges in the maximum matching.
 */
int hopcroftKarp_cuda(BipartiteGraph *graph) {
    // CUDA device memory allocation
    int *d_graph, *d_matchX, *d_matchY, *d_queue, *d_visited, *d_level, *d_m, *d_n;
    hipMalloc((void **)&d_graph, (graph->m + 1) * (graph->n + 1) * sizeof(int));
    hipMalloc((void **)&d_matchX, (graph->m + 1) * sizeof(int));
    hipMalloc((void **)&d_matchY, (graph->n + 1) * sizeof(int));
    hipMalloc((void **)&d_queue, (graph->m + 1) * sizeof(int));
    hipMalloc((void **)&d_visited, (graph->n + 1) * sizeof(int));
    hipMalloc((void **)&d_level, (graph->m + 1) * sizeof(int));
    hipMalloc((void **)&d_m, sizeof(int));
    hipMalloc((void **)&d_n, sizeof(int));

    // Copy graph data to device memory
    hipMemcpy(d_graph, graph, (graph->m + 1) * (graph->n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_m, &(graph->m), sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_n, &(graph->n), sizeof(int), hipMemcpyHostToDevice);

    // Define CUDA kernel parameters
    int block_size = 256;
    int num_blocks = (graph->m + block_size - 1) / block_size;

    // Launch CUDA BFS kernel
    cuda_bfs<<<num_blocks, block_size>>>(d_graph, d_matchX, d_matchY, d_queue, d_visited, d_level, d_m, d_n);

    // Copy matching results back to host memory
    hipMemcpy(graph->matchX, d_matchX, (graph->m + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(graph->matchY, d_matchY, (graph->n + 1) * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_graph);
    hipFree(d_matchX);
    hipFree(d_matchY);
    hipFree(d_queue);
    hipFree(d_visited);
    hipFree(d_level);
    hipFree(d_m);
    hipFree(d_n);

    // Calculate the number of edges in the matching
    int matching = 0;
    for (int u = 1; u <= graph->m; u++) {
        if (graph->matchX[u] != -1)
            matching++;
    }

    return matching;
}